#include "hip/hip_runtime.h"
#include <solana_sdk.h>

__global__ void generate_keys(
    const char *pattern,
    Keypair *results,
    int batch_size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size) return;

    Keypair kp = generate_keypair();
    if (strstr(kp.public_key, pattern) != NULL) {
        results[idx] = kp;
    }
}